
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void saxpy(int n, float a, float * x, float * y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
        y[i] = a * x[i] + y[i];
}

void print(float * a, int n)
{
    for(int i = 0; i < n; ++i)
        printf("%f ", a[i]);
    printf("\n");
}

int main()
{
    float * h_x, * h_y;
    int n;
    float x[] = {1, 1, 1};
    float y[] = {1, 1, 1};
    n = sizeof(x) / sizeof(*x);
    h_x = (float *)malloc(sizeof(x));
    h_y = (float *)malloc(sizeof(y));
    memcpy(h_x, x, sizeof(x));
    memcpy(h_y, y, sizeof(y));
    float * d_x, * d_y;
    int th_per_blk = 256,
    blk_per_grid = (n + th_per_blk - 1) / th_per_blk;
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, n * sizeof(float), hipMemcpyHostToDevice);
    saxpy<<<blk_per_grid, th_per_blk>>>(n, 2.0, d_x, d_y);
    hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
    print(h_y, n);
}
