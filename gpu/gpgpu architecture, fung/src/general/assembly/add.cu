
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int i)
{/* add.type d, a, b; d = a + b */
    int j;
    asm("add.s32 %0, %0, 1;\n" /* \n is used to separate instructions */
        "add.s32 %0, %0, 1;\n" /* ++i */
     : "=r"(i) : "r"(i));
    asm("mov.s32 %0, 0;\n" : "=r"(j)); /* j = 0 */
    printf("i = %d, j = %d\n", i, j);
}

int main()
{
    add<<<1, 1>>>(0);
    hipDeviceSynchronize();
}
